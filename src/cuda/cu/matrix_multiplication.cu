#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <iomanip>

#include <string.h>
#include <math.h>
#include <errno.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <stdlib.h>
#include <unistd.h>

#include <hip/hip_runtime.h>  // For CUDA runtime API
#include <hip/hip_runtime_api.h>  // For checkCudaError macro
#include <helper_timer.h>  // For CUDA SDK timers


#define BLOCK_SIZE 32


/**************************** Logger ********************************/
#define LOG_MESSAGE_SIZE 256

void logger(const char* tag, const char* message) {
   time_t now;
   time(&now);
   printf("%s [%s]: %s\n", ctime(&now), tag, message);
}


void logger_info(const char* message) {
    logger((char *)"INFO",message);
}


void logger_error(const char* message) {
    logger((char *)"ERROR",message);
}


void logger_debug(const char* message) {
    logger((char *)"DEBUG",message);
}

/**********************************************************************/

/**************************** Formatter ********************************/
void getFormattedTime(double seconds, char *formatted_string)
{
    double s(fabs(seconds));
    int h(s/3600);
    int min(s/60 - h*60);
    double sec(s - (h*60 + min)*60);
    std::ostringstream oss;
    oss<<std::setfill('0')<<std::setw(2)<<fabs(seconds)/seconds*h<<":"<<std::setw(2)<<min<<":";
    if (sec/10<1)
        oss<<"0";
    oss<<sec;
    strcpy(formatted_string,oss.str().c_str());
}
/**********************************************************************/


/**************************** FILE I/O ********************************/
#ifndef DATA_DIR
#define DATA_DIR "../../data/"
#endif





static int create_dir(char *rel_path,mode_t mode) {
  return mkdir(rel_path, mode);
}

// Return bytes written in file
int write_matrix_to_file(float *matrix, int rows, int cols, char *matrix_name){
    int ret = 0;
    struct stat st;
    //mode_t mode = st.st_mode & (S_IRUSR | S_IWUSR | S_IXUSR | S_IRGRP | S_IWGRP | S_IXGRP);
    FILE *matrix_file;
    char file_name[63];
    char dir_name[32]; 
    char log_string[LOG_MESSAGE_SIZE];

    sprintf(dir_name,DATA_DIR);

    if ((ret = stat(dir_name, &st)) == -1) { // If directory not found
        create_dir(dir_name,0770); // TODO: Check for the right parametric bitmask
        memset(log_string,0,LOG_MESSAGE_SIZE);
        sprintf(log_string,"Created data directory, with permissions: -rw-r--r-- %s\n",dir_name);
        logger_info(log_string);
    }

    sprintf(&(dir_name)[strlen(dir_name)],"square/matrix%dx%dx%d/",rows,cols,cols);

    if ((ret = stat(dir_name, &st)) == -1) { // If directory not found
        create_dir(dir_name,0770); // TODO: Check for the right parametric bitmask
        memset(log_string,0,LOG_MESSAGE_SIZE);
        sprintf(log_string,"Created data directory, with permissions: -rw-r--r-- %s\n",dir_name);
        logger_info(log_string);
    }

    sprintf(file_name,dir_name);

    sprintf(&(file_name)[strlen(dir_name)],"matrix_%s_%dx%d.bin",matrix_name,rows,cols);

    // Open or create file
	if((matrix_file=fopen(file_name, "w+"))==NULL) {
        memset(log_string,0,LOG_MESSAGE_SIZE);
        sprintf(log_string,"Error opening file %s\n",file_name);
		logger_info(log_string);
		exit(EXIT_FAILURE);
	}

    // Write rows number as first element of the binary file
    ret = fwrite(&rows,sizeof(int),1,matrix_file);
    if(ret == 0){
        memset(log_string,0,LOG_MESSAGE_SIZE);
        sprintf(log_string,"Error writing bytes on file %s\n",file_name);
		logger_info(log_string);
        exit(EXIT_FAILURE);
    }
    
    // Write cols number as second element of the binary file
    ret = fwrite(&cols,sizeof(int),1,matrix_file);
    if(ret == 0){
        memset(log_string,0,LOG_MESSAGE_SIZE);
        sprintf(log_string,"Error opening file %s\n",file_name);
		logger_info(log_string);
        exit(EXIT_FAILURE);
    }

    ret = fwrite(matrix,sizeof(float),rows*cols,matrix_file);
    if(ret == 0){
        memset(log_string,0,LOG_MESSAGE_SIZE);
        sprintf(log_string,"Error opening file %s\n",file_name);
		logger_info(log_string);
        exit(EXIT_FAILURE);
    }
    

    fclose(matrix_file);


    memset(log_string,0,LOG_MESSAGE_SIZE);
    sprintf(log_string,"File %s populated correctly!\n",file_name);
	logger_info(log_string);

    return sizeof(float)*cols*rows + sizeof(int) + sizeof(int);
}


void read_matrix_from_file(float *matrix,int rows_expected, int cols_expected, char *matrix_name){
    FILE *matrix_file;
    char file_name[64];
    char log_string[LOG_MESSAGE_SIZE];
    int rows;
    int cols;
    int ret = 0;

    sprintf(file_name,DATA_DIR);
    sprintf(&(file_name)[strlen(file_name)],"square/matrix%dx%dx%d/matrix_%s_%dx%d.bin",rows_expected,rows_expected,cols_expected,matrix_name,rows_expected,cols_expected);

    // Open or create file
	  if((matrix_file=fopen(file_name, "r"))==NULL) {
        memset(log_string,0,LOG_MESSAGE_SIZE);
        sprintf(log_string,"Error opening file %s\n",file_name);
		  logger_error(log_string);
		  exit(EXIT_FAILURE);
	  }

    // Read rows number as first element of the binary file
    ret = fread(&rows,sizeof(int),1,matrix_file);
    if(ret == 0){
        memset(log_string,0,LOG_MESSAGE_SIZE);
        sprintf(log_string,"Error writing bytes on file %s\n",file_name);
        logger_error(log_string); //TODO: Check with errno
        exit(EXIT_FAILURE);
    }else if(rows != rows_expected){    
        logger_error("Matrix passed to the function must be choerent with the size of the matrix in the file!\n");
        exit(EXIT_FAILURE);
    }

    // Read cols number as first element of the binary file
    ret = fread(&cols,sizeof(int),1,matrix_file);
    if(ret == 0){
        memset(log_string,0,LOG_MESSAGE_SIZE);
        sprintf(log_string,"Error writing bytes on file %s\n",file_name);
        logger_error(log_string); //TODO: Check with errno
        exit(EXIT_FAILURE);
    }else if(cols != cols_expected){
        logger_error("Matrix passed to the function must be choerent with the size of the matrix in the file!\n");
        exit(EXIT_FAILURE);
    }


    ret = fread(matrix,sizeof(float),rows*cols,matrix_file);
    if(ret == 0 && errno == EOF){
        memset(log_string,0,LOG_MESSAGE_SIZE);
        sprintf(log_string,"Error reading bytes from file %s to matrix! Error returned: %s\n",file_name, strerror(errno));
        logger_error(log_string); 
        exit(EXIT_FAILURE);
    }
    


    fclose(matrix_file);
}


void write_cuda_stats(int N, int K, int M,double total_time,float max_diff,float max_rel_diff,float gflops){
    int ret = 0;
    struct stat st;
    FILE *matrix_file;
    char *file_name = (char *)"data/stats.csv";
    char *dir_name = (char *)"data/"; 
    char log_string[LOG_MESSAGE_SIZE];

    // Create directory if needed
    if ((ret = stat(dir_name, &st)) == -1) { // If directory not found
        create_dir(dir_name,0770); 
        memset(log_string,0,LOG_MESSAGE_SIZE);
        sprintf(log_string,"Created data directory, with permissions: -rw-r--r-- %s\n",dir_name);
        logger_info(log_string);
    }


    // Open or create file
	  if((matrix_file=fopen(file_name, "a"))==NULL) {
      memset(log_string,0,LOG_MESSAGE_SIZE);
      sprintf(log_string,"Error opening file %s\n",file_name);
		  logger_info(log_string);
		  exit(EXIT_FAILURE);
	  } 

    fprintf(matrix_file,"%d,%d,%d,%f,%f,%f,%f\n",N,K,M,total_time,max_diff,max_rel_diff,gflops);

    fclose(matrix_file);
}


#ifdef DEBUG


/**********************************************************************/

/**
 * @brief
 * @param matrix
 * @param rows 
 * @param cols
*/
void printMatrix(const float *matrix,int rows,int cols){
  std::cout << "______________________________________________________________________________________________________________________________________________________\n";
  for (int row = 0; row < rows; ++row) {
    std::cout << "| ";
    for (int col = 0; col < cols; ++col) {
      int idx = row * cols + col;
      std::cout << matrix[idx] << " ";
    }
    std::cout << "\t\t|\n";
  }
  std::cout << "______________________________________________________________________________________________________________________________________________________\n\n";
}


/**
 * @brief An implementation of the sequential matrix by matrix multiplication, taking in count memory acces
 * and compiled with -O3. It's used only for debugging and for the developement phase of the project.
 * @param matrix_A The memory in which are the values stored in the matrix A N x K
 * @param matrix_B The memory in which are the values stored in the matrix B K x M
 * @param matrix_C The memory in which are the values stored in the matrix C N x M
 * @param N Rows dimwnsion of the matrix A and C
 * @param K Colums dimension of the matrix A and rows dimension of the matrix B
 * @param M Columns dimension of the matrix B and C
*/
void CpuMatrixVector(float *matrix_A, float *matrix_B, float *matrix_C, int N, int K, int M) {
    for (int i = 0; i < N; ++i) 
    {
        for (int h = 0; h < K; ++h)
        {
            for (int j = 0; j < M; ++j) 
            {
                matrix_C[i * M + j] += matrix_A[i * K + h] * matrix_B[h * M + j];
            }
        }
    }

}
#endif


__global__ void gpuMatrixVectorSharedMemory(float* A, float* B, float* C, int N, int K, int M)
{
    __shared__ float matrix_A_shared[BLOCK_SIZE][BLOCK_SIZE]; // Memory in which is stored the matrix A shared between all the threads in the tile
    __shared__ float matrix_B_shared[BLOCK_SIZE][BLOCK_SIZE];
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    float sum = 0;
    int num_blocks = (int) (N/BLOCK_SIZE)

    for(int i=0; i<num_blocks; ++i)
    {
        // Load data in the tile of shared memory
        if(row<M && t*BLOCK_SIZE+tx<N)
            matrix_A_shared[ty][tx] = A[row*N + i*BLOCK_SIZE+tx];
        else
            matrix_A_shared[ty][tx] = 0.0;
        if(i*BLOCK_SIZE+ty<N && col<K)
            matrix_B_shared[ty][tx] = B[(t*BLOCK_SIZE+ty)*K + col];
        else
            matrix_B_shared[ty][tx] = 0.0;
        __syncthreads();

        // Actual computation over the single tile
        for(int j=0; j<BLOCK_SIZE; ++j)
            sum += matrix_A_shared[ty][j] * matrix_B_shared[j][tx];
        __syncthreads();
    }
    if(row<M && col<K)
        C[col+row*K] += sum;

}



__global__ void gpuMatrixVector(float* A, float* B, float* C, int N, int K, int M)
{
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  
  if (row < N && col < M) {
    for (int i = 0; i < K; ++i) {
      C[row * M + col]  += A[row * K + i] * B[i * M + col];
      }
  }

}

int main(int argc, char** argv) {
  int N = 0;
  int K = 0;
  int M = 0;

	if(argc != 4){
        logger_error("Program should be called like ./<elf file name> <N> <K> <M>");
        exit(EXIT_FAILURE);
    }

    if (sscanf (argv[1], "%d", &N) != 1) {
        fprintf(stderr, "[ERROR] - not an integer");
        logger_error("Program should be called like ./<elf file name> <N> <K> <M>");
        exit(EXIT_FAILURE);
    }

    if (sscanf (argv[2], "%d", &K) != 1) {
        fprintf(stderr, "[ERROR] - not an integer");
        logger_error("Program should be called like ./<elf file name> <N> <K> <M>");
        exit(EXIT_FAILURE);
    }

	if (sscanf (argv[3], "%d", &M) != 1) {
        fprintf(stderr, "[ERROR] - not an integer");
        logger_error("Program should be called like ./<elf file name> <N> <K> <M>");
        exit(EXIT_FAILURE);
    }


  
  // ----------------------- Host memory initialisation ----------------------- //
  float* h_A = new float[N * K];
  float* h_B = new float[K * M];
  float* h_C = new float[N * M];

  float* h_C_result_host = new float[N * M];
  float* h_C_result_device = new float[N * M];

  float time;
  char formatted_time_string[64];
  hipEvent_t start, stop;

  dim3 blockDim(BLOCK_SIZE,BLOCK_SIZE); // Don't need to write z = 1, max 1024
  int gx = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;//(m % blockDim.x==0) ? m / blockDim.x : m / blockDim.x + 1;
  int gy = (M + BLOCK_SIZE - 1) / BLOCK_SIZE;//(n % blockDim.y==0) ? n / blockDim.y : n / blockDim.y + 1;
  dim3 gridDim(gx, gy);
  printf("%d %d\n",gx,gy);

  read_matrix_from_file(h_A,N,K,(char *)"A");
  read_matrix_from_file(h_B,K,M,(char *)"B");
  read_matrix_from_file(h_C,N,M,(char *)"C");
  read_matrix_from_file(h_C_result_host,N,M,(char *)"sequential_C");

  std::cout << "[INFO] Matrix initialization done.\n";

#ifdef DEBUG
  puts("\t[MATRIX] Matrix A");
  printMatrix(h_A,N,K);
  puts("\t[MATRIX] Matrix B");
  printMatrix(h_B,K,M);
  puts("\t[MATRIX] Matrix C");
  printMatrix(h_C,N,M);
#endif
  std::cout << "Test case: " << N  << " x " << M << std::endl;
// ---------------------- Device memory initialisation ---------------------- //

  float *d_A, *d_B, *d_C;

  checkCudaErrors(hipMalloc((void**) &d_A, N * K * sizeof(float)));
  checkCudaErrors(hipMalloc((void**) &d_B, K * M * sizeof(float)));
  checkCudaErrors(hipMalloc((void**) &d_C, N * M * sizeof(float)));

  // Copy matrices from the host (CPU) to the device (GPU).
  checkCudaErrors(hipMemcpy(d_A, h_A, N * K * sizeof(float), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_B, h_B, K * M * sizeof(float), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_C, h_C, N * M * sizeof(float), hipMemcpyHostToDevice));

#ifdef DEBUG
  // ------------------------ Calculations on the CPU ------------------------- //

  // Create the CUDA SDK timer.
  StopWatchInterface* timer = 0;
  sdkCreateTimer(&timer);

  timer->start();
  
  CpuMatrixVector(h_A, h_B, h_C_result_host, N, K ,M);

  timer->stop();
  double cpuflops=flopcnt/ timer->getTime();
  std::cout << "  CPU time: " << timer->getTime() << " ms." << " MFLOPS " << cpuflops << std::endl;

  puts("\t[RESULT] Sequential result");
  printMatrix(h_C_result_host,N,M);
#endif

// ------------------------ Calculations on the GPU ------------------------- //
  
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);


  gpuMatrixVector<<<gridDim, blockDim >>>(d_A, d_B, d_C, N, K, M);
  checkCudaErrors(hipDeviceSynchronize());

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);

  std::cout << "Time to generate: " << time << " ms\n";


  double gpuflops= 2.0*N*K*M / time;
  memset(formatted_time_string, 0, strlen(formatted_time_string));
  getFormattedTime(time/1000.0f,formatted_time_string);
  std::cout << "  GPU time: " << formatted_time_string << " MFLOPS " << gpuflops<<std::endl;

  // Download the resulting vector d_y from the device and store it in h_y_d.
  checkCudaErrors(hipMemcpy(h_C_result_device, d_C, N * M * sizeof(float),hipMemcpyDeviceToHost));

#ifdef DEBUG
  puts("\t[RESULT] CUDA result");
  printMatrix(h_C_result_device,N,M);
#endif
  // Now let's check if the results are the same.
  float reldiff = 0.0f;
  float diff = 0.0f;
  
  for (int row = 0; row < N; ++row) {
    for(int col = 0; col < M; ++col){
      float maxabs = std::max(std::abs(h_C_result_device[row * M + col]),std::abs(h_C_result_host[row * M + col]));
      if (maxabs == 0.0){
        maxabs=1.0;
      } 
      reldiff = std::max(reldiff, std::abs(h_C_result_device[row * M + col] - h_C_result_host[row * M + col])/maxabs);
      diff = std::max(diff, std::abs(h_C_result_device[row * M + col] - h_C_result_host[row * M + col]));
    }
  }
  std::cout << "Max diff =  " << diff << "  Max rel diff = " << reldiff << std::endl;
  fflush(stdout);
  write_cuda_stats(N, K, M,time,diff,reldiff,gflops);


// ------------------------------- Cleaning up ------------------------------ //

  write_matrix_to_file(h_C_result_device,N,M,(char *)"cuda_C");
#ifdef DEBUG
  delete timer;
#endif
  checkCudaErrors(hipFree(d_A));
  checkCudaErrors(hipFree(d_B));
  checkCudaErrors(hipFree(d_C));

  delete[] h_A;
  delete[] h_B;
  delete[] h_C;
  delete[] h_C_result_host;
  delete[] h_C_result_device;
  return 0;
}
