#include "hip/hip_runtime.h"
// i.e. A[i, j] is stored in i * ncols + j element of the vector.
//

#include <iostream>

#include <hip/hip_runtime.h>  // For CUDA runtime API
#include <hip/hip_runtime_api.h>  // For checkCudaError macro
#include <helper_timer.h>  // For CUDA SDK timers

#define BLOCK_SIZE 32

void printMatrix(const float *h_A,int ROWS,int COLS){
  std::cout << "______________________________________________________________________________________________________________________________________________________\n";
  for (int row = 0; row < ROWS; ++row) {
    std::cout << "| ";
    for (int col = 0; col < COLS; ++col) {
      int idx = row * COLS + col;
      std::cout << h_A[idx] << " ";
    }
    std::cout << "\t\t|\n";
  }
  std::cout << "______________________________________________________________________________________________________________________________________________________\n\n";
}



// Simple CPU implementation of matrix addition.
void CpuMatrixVector(float *matrix_A, float *matrix_B, float *matrix_C, int N, int K, int M) {
    for (int i = 0; i < N; ++i) 
    {
        for (int h = 0; h < K; ++h)
        {
            for (int j = 0; j < M; ++j) 
            {
                matrix_C[i * M + j] += matrix_A[i * K + h] * matrix_B[h * M + j];
            }
        }
    }

}

__global__ void gpuMatrixVector(float* A, float* B, float* C, int n, int k, int m)
{
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  
  if (row < n && col < m) {
    for (int i = 0; i < k; ++i) {
      C[row * m + col]  += A[row * k + i] * B[i * m + col];
      }
  }

}


int main(int argc, char** argv) {

  if (argc != 4) {
    fprintf(stderr,"[ERROR] Correct usage: %s n k m\nYou should insert matrices dimensons!\n",argv[0]);
    exit(EXIT_FAILURE);
  }
  int n = (int) atoi(argv[1]);
  int k = (int) atoi(argv[2]);
  int m = (int) atoi(argv[3]);

  
  
  // ----------------------- Host memory initialisation ----------------------- //

  float* h_A = new float[n * k];
  float* h_B = new float[k * m];
  float* h_C = new float[n * m];

  float* h_C_result_host = new float[n * m];
  float* h_C_result_device = new float[n * m];

  float time;
  hipEvent_t start, stop;

  dim3 blockDim(BLOCK_SIZE,BLOCK_SIZE); // Don't need to write z = 1, max 1024
  int gx = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;//(m % blockDim.x==0) ? m / blockDim.x : m / blockDim.x + 1;
  int gy = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;//(n % blockDim.y==0) ? n / blockDim.y : n / blockDim.y + 1;
  dim3 gridDim(gx, gy);
  printf("%d %d\n",gx,gy);

  srand(123456);
  for (int row = 0; row < n; ++row) {
    for (int col = 0; col < m; ++col) {
      int idx = row * m + col;
      h_C[idx] = 100.0f * static_cast<float>(rand()) / RAND_MAX;
    }
  }


  for (int row = 0; row < n; ++row) {
    for (int col = 0; col < k; ++col) {
      int idx = row * k + col;
      h_A[idx] = 100.0f * static_cast<float>(rand()) / RAND_MAX;
    }
  }

  for (int row = 0; row < k; ++row) {
    for (int col = 0; col < m; ++col) {
      int idx = row * m + col;
      h_B[idx] = 100.0f * static_cast<float>(rand()) / RAND_MAX;
    }
  }

  std::cout << "[INFO] Matrix initialization done.\n";

  memcpy(h_C_result_host,h_C,n * m * sizeof(float));
#ifdef DEBUG
  puts("\t[MATRIX] Matrix A");
  printMatrix(h_A,n,k);
  puts("\t[MATRIX] Matrix B");
  printMatrix(h_B,k,m);
  puts("\t[MATRIX] Matrix C");
  printMatrix(h_C,n,m);
#endif
  std::cout << "Test case: " << n  << " x " << m << std::endl;
// ---------------------- Device memory initialisation ---------------------- //

  float *d_A, *d_B, *d_C;

  checkCudaErrors(hipMalloc((void**) &d_A, n * k * sizeof(float)));
  checkCudaErrors(hipMalloc((void**) &d_B, k * m * sizeof(float)));
  checkCudaErrors(hipMalloc((void**) &d_C, n * m * sizeof(float)));

  // Copy matrices from the host (CPU) to the device (GPU).
  checkCudaErrors(hipMemcpy(d_A, h_A, n * k * sizeof(float), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_B, h_B, k * m * sizeof(float), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_C, h_C, n * m * sizeof(float), hipMemcpyHostToDevice));

  // ------------------------ Calculations on the CPU ------------------------- //
  float flopcnt=2.e-3*n*m;
  
  // Create the CUDA SDK timer.
  StopWatchInterface* timer = 0;
  sdkCreateTimer(&timer);

  timer->start();
  
  CpuMatrixVector(h_A, h_B, h_C_result_host, n, k ,m);

  timer->stop();
  float cpuflops=flopcnt/ timer->getTime();
  std::cout << "  CPU time: " << timer->getTime() << " ms." << " MFLOPS " << cpuflops << std::endl;

#ifdef DEBUG
  puts("\t[RESULT] Sequential result");
  printMatrix(h_C_result_host,n,m);
#endif

// ------------------------ Calculations on the GPU ------------------------- //

  // TODO Calculate the dimension of the grid of blocks. A 1D grid suffices.
  
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);


  gpuMatrixVector<<<gridDim, blockDim >>>(d_A, d_B, d_C, n, k, m);
  checkCudaErrors(hipDeviceSynchronize());

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);

  std::cout << "Time to generate: " << time << " ms\n";


  float gpuflops=flopcnt/ time;
  std::cout << "  GPU time: " << time << " ms." << " MFLOPS " << gpuflops<<std::endl;

  // Download the resulting vector d_y from the device and store it in h_y_d.
  checkCudaErrors(hipMemcpy(h_C_result_device, d_C, n * m * sizeof(float),hipMemcpyDeviceToHost));

#ifdef DEBUG
  puts("\t[RESULT] CUDA result");
  printMatrix(h_C_result_device,n,m);
#endif
  // Now let's check if the results are the same.
  float reldiff = 0.0f;
  float diff = 0.0f;
  
  for (int row = 0; row < n; ++row) {
    for(int col = 0; col < m; ++col){
      float maxabs = std::max(std::abs(h_C_result_device[row * m + col]),std::abs(h_C_result_host[row * m + col]));
      if (maxabs == 0.0){
        maxabs=1.0;
      } 
      reldiff = std::max(reldiff, std::abs(h_C_result_device[row * m + col] - h_C_result_host[row * m + col])/maxabs);
      diff = std::max(diff, std::abs(h_C_result_device[row * m + col] - h_C_result_host[row * m + col]));
    }
  }
  std::cout << "Max diff =  " << diff << "  Max rel diff = " << reldiff << std::endl;
  fflush(stdout);
  // Rel diff should be as close as possible to unit roundoff; float
  // corresponds to IEEE single precision, so unit roundoff is
  // 1.19e-07
  // 

// ------------------------------- Cleaning up ------------------------------ //

  delete timer;

  checkCudaErrors(hipFree(d_A));
  checkCudaErrors(hipFree(d_B));
  checkCudaErrors(hipFree(d_C));

  delete[] h_A;
  delete[] h_B;
  delete[] h_C;
  delete[] h_C_result_host;
  delete[] h_C_result_device;
  return 0;
}
