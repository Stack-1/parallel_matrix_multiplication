#include "hip/hip_runtime.h"
// i.e. A[i, j] is stored in i * ncols + j element of the vector.
//

#include <iostream>

#include <hip/hip_runtime.h>  // For CUDA runtime API
#include <hip/hip_runtime_api.h>  // For checkCudaError macro
#include <helper_timer.h>  // For CUDA SDK timers


// TODO What is a good initial guess for XBD and YBD (both
// greater than 1) ?
// After you get the code to work, experiment with different sizes
// to find the best possible performance
// Note: For meaningful time measurements you need sufficiently large matrices.
#define XBD 16
#define YBD 16

const int TILE_WIDTH = 32;

const dim3 BLOCK_DIM(TILE_WIDTH, TILE_WIDTH, 1);




void printMatrix(const float *h_A,int ROWS,int COLS){
  std::cout << "______________________________________________________________________________________________________________________________________________________\n";
  for (int row = 0; row < ROWS; ++row) {
    std::cout << "| ";
    for (int col = 0; col < COLS; ++col) {
      int idx = row * COLS + col;
      std::cout << h_A[idx] << " ";
    }
    std::cout << "\t\t|\n";
  }
  std::cout << "______________________________________________________________________________________________________________________________________________________\n\n";
}



// Simple CPU implementation of matrix addition.
void CpuMatrixVector(float *matrix_A, float *matrix_B, float *matrix_C, size_t N, size_t K, size_t M) {
  for (size_t i=0;i<N;i++) 
        for(size_t j=0;j<M;j++) {
            for(size_t k=0;k<K;k++){
                matrix_C[i * M + j]=matrix_C[i * M + j]+matrix_A[i * K + k]*matrix_B[k * M + j];
			}
    }
}

__global__ void gpuMatrixVector(float* A_d, float* B_d, float* C_d, int m, int k, int n)
{
    __shared__ float ds_A[TILE_WIDTH][TILE_WIDTH];
    __shared__ float ds_B[TILE_WIDTH][TILE_WIDTH];
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    float sum = 0;

    for(int t=0; t<(n-1)/TILE_WIDTH+1; t++)
    {
        if(row<m && t*TILE_WIDTH+tx<n)
            ds_A[ty][tx] = A_d[row*n + t*TILE_WIDTH+tx];
        else
            ds_A[ty][tx] = 0.0;
        if(t*TILE_WIDTH+ty<n && col<k)
            ds_B[ty][tx] = B_d[(t*TILE_WIDTH+ty)*k + col];
        else
            ds_B[ty][tx] = 0.0;
        __syncthreads();
        for(int i=0; i<TILE_WIDTH; i++)
            sum += ds_A[ty][i] * ds_B[i][tx];
        __syncthreads();
    }
    if(row<m && col<k)
        C_d[col+row*k] += sum;
}


int main(int argc, char** argv) {

  if (argc != 4) {
    fprintf(stderr,"[ERROR] Correct usage: %s n k m\nYou should insert matrices dimensons!\n",argv[0]);
    exit(EXIT_FAILURE);
  }
  size_t n = (size_t) atoi(argv[1]);
  size_t k = (size_t) atoi(argv[2]);
  size_t m = (size_t) atoi(argv[3]);

  
  
  // ----------------------- Host memory initialisation ----------------------- //

  float* h_A = new float[n * k];
  float* h_B = new float[k * m];
  float* h_C = new float[n * m];

  float* h_C_result_host = new float[n * m];
  float* h_C_result_device = new float[n * m];

  srand(123456);
  for (size_t row = 0; row < n; ++row) {
    for (size_t col = 0; col < m; ++col) {
      size_t idx = row * m + col;
      h_A[idx] = 100.0f * static_cast<float>(rand()) / RAND_MAX;
      h_B[idx] = 100.0f * static_cast<float>(rand()) / RAND_MAX;
      h_C[idx] = 100.0f * static_cast<float>(rand()) / RAND_MAX;
    }
  }

  memcpy(h_C_result_host,h_C,n * m * sizeof(float));

/*
  puts("\t[MATRIX] Matrix A");
  printMatrix(h_A,n,k);
  puts("\t[MATRIX] Matrix B");
  printMatrix(h_B,k,m);
  puts("\t[MATRIX] Matrix C");
  printMatrix(h_C,n,m);
  puts("\t[MATRIX] Matrix h_C_result_host");
  printMatrix(h_C_result_host,n,m);*/

  std::cout << "Test case: " << n  << " x " << m << std::endl;
// ---------------------- Device memory initialisation ---------------------- //

  float *d_A, *d_B, *d_C;

  checkCudaErrors(hipMalloc((void**) &d_A, n * k * sizeof(float)));
  checkCudaErrors(hipMalloc((void**) &d_B, k * m * sizeof(float)));
  checkCudaErrors(hipMalloc((void**) &d_C, n * m * sizeof(float)));

  // Copy matrices from the host (CPU) to the device (GPU).
  checkCudaErrors(hipMemcpy(d_A, h_A, n * k * sizeof(float), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_B, h_B, k * m * sizeof(float), hipMemcpyHostToDevice));

  // ------------------------ Calculations on the CPU ------------------------- //
  float flopcnt=2.e-6*n*m;
  
  // Create the CUDA SDK timer.
  StopWatchInterface* timer = 0;
  sdkCreateTimer(&timer);

  timer->start();
  CpuMatrixVector(h_A, h_B, h_C_result_host, n, k ,m);

  timer->stop();
  float cpuflops=flopcnt/ timer->getTime();
  std::cout << "  CPU time: " << timer->getTime() << " ms." << " GFLOPS " << cpuflops << std::endl;
/*
  puts("\t[RESULT] Sequential result");
  printMatrix(h_C_result_host,n,m);*/


// ------------------------ Calculations on the GPU ------------------------- //

  // TODO Calculate the dimension of the grid of blocks. A 1D grid suffices.
  
  dim3 GRID_DIM((k-1)/TILE_WIDTH+1, (m-1)/TILE_WIDTH+1, 1);

  timer->reset();
  timer->start();
  gpuMatrixVector<<<GRID_DIM, BLOCK_DIM >>>(d_A, d_B, d_C, n, m, k);
  checkCudaErrors(hipDeviceSynchronize());

  timer->stop();
  float gpuflops=flopcnt/ timer->getTime();
  std::cout << "  GPU time: " << timer->getTime() << " ms." << " GFLOPS " << gpuflops<<std::endl;

  // Download the resulting vector d_y from the device and store it in h_y_d.
  checkCudaErrors(hipMemcpy(h_C_result_device, d_C, n * m * sizeof(float),hipMemcpyDeviceToHost));
/*
  puts("\t[RESULT] CUDA result");
  printMatrix(h_C_result_device,n,m);*/

  // Now let's check if the results are the same.
  float reldiff = 0.0f;
  float diff = 0.0f;
  
  for (int row = 0; row < n; ++row) {
    for(int col = 0; col < m; ++col){
      float maxabs = std::max(std::abs(h_C_result_device[row * m + col]),std::abs(h_C_result_host[row * m + col]));
      if (maxabs == 0.0){
        maxabs=1.0;
      } 
      reldiff = std::max(reldiff, std::abs(h_C_result_device[row * m + col] - h_C_result_host[row * m + col])/maxabs);
      diff = std::max(diff, std::abs(h_C_result_device[row * m + col] - h_C_result_host[row * m + col]));
    }
  }
  std::cout << "Max diff = " << diff << "  Max rel diff = " << reldiff << std::endl;
  // Rel diff should be as close as possible to unit roundoff; float
  // corresponds to IEEE single precision, so unit roundoff is
  // 1.19e-07
  // 

// ------------------------------- Cleaning up ------------------------------ //

  delete timer;

  checkCudaErrors(hipFree(d_A));
  checkCudaErrors(hipFree(d_B));
  checkCudaErrors(hipFree(d_C));

  //delete[] h_A;
  //delete[] h_B;
  //delete[] h_C;
  //delete[] h_C_result_host;
  //delete[] h_C_result_device;
  return 0;
}
